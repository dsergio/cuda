


#include <hip/hip_runtime.h>
#include <stdio.h>

// test function, do nothing
__global__ void test_kernel(void) {
	printf("do nothing\n");
}

// add integers:
__global__ void add(int *a, int *b, int *c) {
	*c = *a + *b;
	printf("IN ADD: a = %i\n", *a);
}

int main(void) {
	printf("test CUDA\n\n");

	test_kernel<<<1,1>>>();

	int a, b, c;

	int *d_a, *d_b, *d_c;

	int size = sizeof(int);

	hipMalloc((void **) &d_a, size);
	hipMalloc((void **) &d_b, size);
	hipMalloc((void **) &d_c, size);

	a = 2;
	b = 2;

	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);
	hipMemcpy(d_c, &c, size, hipMemcpyHostToDevice);

	add<<<1,1>>>(d_a, d_b, d_c);

	hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

	printf("a = %i\nb = %i\n\na + b = c\n\nc = %i\n\n", a, b, c);
	printf("d_a = %p\n", d_a);
	printf("d_b = %p\n", d_b);
	printf("d_c = %p\n", d_c);

	printf("\n");
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	return 0;
}

