


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <math.h>
#include <iomanip>
#include <chrono>

using namespace std;



#define _USE_MATH_DEFINES


#define THREADS_PER_BLOCK 32
#define N 4


struct location_item {
	const char * name;
	float lat;
	float lon;
};

__device__ float toRad(float degree) {
    return degree / 180 * M_PI;
}

__device__ float calculateDistance(float lat1, float long1, float lat2, float long2) {

	if (lat1 == lat2 && long1 == long2) {
		return 0;
	}
    float dist;
    dist = sin(toRad(lat1)) * sin(toRad(lat2)) + cos(toRad(lat1)) * cos(toRad(lat2)) * cos(toRad(long1 - long2));
    dist = acos(dist);

		// dist = (6371 * pi * dist) / 180;
		// got dist in radian, no need to change back to degree and convert to rad again.

    dist = 6371 * dist;

    return dist;
}

// test function, do nothing
__global__ void test_kernel(void) {
	printf("do nothing\n");
}

// add integers:
__global__ void add(int *a, int *b, int *c) {
	*c = *a + *b;
	printf("IN ADD: a = %i\n", *a);
}

// calculate_distance:
__global__ void calculate_distance(float *lat, float *lon, float *dist, int n) {

	int i = threadIdx.x + blockIdx.x * blockDim.x;

	int i_m = ((n + 1) * (n + 1) - 1);

	// printf("i: %d, n: %d, i_m: %d\n", i, n, i_m);
	if (i <= i_m) {

		int i_src = i / (n + 1);
		int i_dest = i % (n + 1);

		float d = calculateDistance(lat[i_src], lon[i_src], lat[i_dest], lon[i_dest]);
		d = round(d);

		// printf("i: %d, n: %d, i_m: %d, i_src: %d, i_dest: %d " 
		// 	"lat src: %lf, lon src: %lf, lat dest: %lf, lon dest: %lf, d=%lf\n", 
		// 	i, n, i_m, i_src, i_dest, lat[i_src], lon[i_src], lat[i_dest], lon[i_dest], d);

		

		dist[i] = d;
	}
	
}

auto print_time() {
	auto now = chrono::system_clock::now();

    // Convert the current time to time since epoch
    auto duration = now.time_since_epoch();

    // Convert duration to milliseconds
    auto milliseconds
        = chrono::duration_cast<chrono::milliseconds>(
              duration)
              .count();

    // Print the result
    cout << "Current time in milliseconds is: "
         << milliseconds << endl;
    return milliseconds;
}


int main(void) {

	printf("test CUDA\n\n");
	
	
	auto start_time = print_time();

	// location_item items[N];

	const long long unsigned coords_size = (N) * sizeof(float);
	const long long unsigned i_M = ((N) * (N) - 1);
	const long long unsigned coords_size_distance = i_M * sizeof(float);

	cout << "N : " << N << ", i_M: " << i_M << endl;

	cout << "coords_size: " << coords_size << 
		", coords_size_distance: " << coords_size_distance << ", INT_MAX: " << INT_MAX << endl;
	
	float *lat_arr, *lon_arr, *dist_arr;
	lat_arr = (float *) malloc(coords_size);
	lon_arr = (float *) malloc(coords_size);
	dist_arr = (float *) malloc(coords_size_distance);

	float *d_lat_arr, *d_lon_arr, *d_dist_arr;


	string names_arr[N + 1];

	// load data line by line O(n)...
	int i = 0;
	string line;
	ifstream myfile ("wa_cities");
	if (myfile.is_open()) {
		while (getline(myfile, line) && i < N) {
			
			int pos = line.find(";");
			string name = line.substr(0, pos);
			string coords = line.substr(pos + 1, strlen(line.c_str()) - pos);
			int comma = coords.find(",");
			float lat = stod(coords.substr(0, comma));
			float lon = stod(coords.substr(comma + 1, strlen(coords.c_str()) - comma));

			// items[i] = (location_item){name.c_str(), lat, lon};
			lat_arr[i] = lat;
			lon_arr[i] = lon;

			names_arr[i] = name;
			i++;
		}
		myfile.close();
	} else cout << "Unable to open file";



	// for (int i = 0; i < N; i++) {
	// 	printf("i = %d: before CUDA: lat: %lf, name: %s\n", i, lat_arr[i], names_arr[i].c_str());
	// }

	cout << "blocks: " << i_M/THREADS_PER_BLOCK + 1<< ", threads per block: " << THREADS_PER_BLOCK << endl;

	// CUDA:
	
	hipMalloc((void **) &d_lat_arr, coords_size);
	hipMalloc((void **) &d_lon_arr, coords_size);
	hipMalloc((void **) &d_dist_arr, coords_size_distance);

	hipMemcpy(d_lat_arr, lat_arr, coords_size, hipMemcpyHostToDevice);
	hipMemcpy(d_lon_arr, lon_arr, coords_size, hipMemcpyHostToDevice);

	calculate_distance<<<i_M/THREADS_PER_BLOCK + 1,THREADS_PER_BLOCK>>>(d_lat_arr, d_lon_arr, d_dist_arr, N - 1);


	hipError_t error = hipGetLastError();
	if(error != hipSuccess)
	{
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
		exit(-1);
	}

	hipMemcpy(dist_arr, d_dist_arr, coords_size_distance, hipMemcpyDeviceToHost);


	ofstream output_file;
  	output_file.open ("distances_matrix", ios::trunc);


	for (int i = 0; i <= i_M; i++) {

		int i_src = i / (N);
		int i_dest = i % (N);

		// if (true) { // (dist_arr[i] > 500) {
		// 	cout << "after CUDA:" << "i_src: " << i_src << ", i_dest: " << i_dest << endl
		// 	<< "\tlat src: " <<  lat_arr[i_src] << ", lon src: " << lon_arr[i_src] << endl
		// 	<< "\tlat dest: " <<  lat_arr[i_dest] << ", lon dest: " << lon_arr[i_dest] << endl
		// 	<< "\tdistance (from " << names_arr[i_src] << " to " << names_arr[i_dest] << "): \t\t"
		// 	<< setprecision(3) << dist_arr[i] << endl;
		// }
		

		if (i > 0 && i % N == 0) {
			output_file << endl;
			output_file << dist_arr[i];
		} else {
			if (i > 0) {
				output_file << " ";
			}
			output_file << dist_arr[i];
		}
	}

	output_file.close();

	free(lat_arr);
	free(lon_arr);
	free(dist_arr);
	hipFree(d_lat_arr);
	hipFree(d_lon_arr);
	hipFree(d_dist_arr);

	auto end_time = print_time();
	cout << "time in milliseconds: " << end_time - start_time << endl;

	return 0;
}

