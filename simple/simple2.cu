

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <math.h>

using namespace std;

#define _USE_MATH_DEFINES

struct location_item {
	string name;
	double lat;
	double lon;
};

// calculate_distance:
__global__ void calculate_distance(location_item **items) {
	printf("items:");
}

// test function, do nothing
__global__ void test_kernel(void) {
	printf("do nothing\n");
}


double toRad(double degree) {
    return degree / 180 * M_PI;
}

double calculateDistance(double lat1, double long1, double lat2, double long2) {
    double dist;
    dist = sin(toRad(lat1)) * sin(toRad(lat2)) + cos(toRad(lat1)) * cos(toRad(lat2)) * cos(toRad(long1 - long2));
    dist = acos(dist);

		// dist = (6371 * pi * dist) / 180;
		// got dist in radian, no need to change back to degree and convert to rad again.

    dist = 6371 * dist;

    return dist;
}


int main(void) {

	printf("test CUDA\n\n");

	// load data line by line O(n)... can this be parallelized?
	
	const int SIZE = 1000;

	location_item items[SIZE];
	location_item *d_items[SIZE];

	int i = 0;
	string line;
	ifstream myfile ("wa_cities");
	if (myfile.is_open()) {
		while (getline(myfile, line)) {
			
			int pos = line.find(";");
			string name = line.substr(0, pos);
			string coords = line.substr(pos + 1, strlen(line.c_str()) - pos);
			int comma = coords.find(",");
			double lat = stod(coords.substr(0, comma));
			double lon = stod(coords.substr(comma + 1, strlen(coords.c_str()) - comma));

			items[i] = (location_item){name, lat, lon};

			i++;
		}
		myfile.close();
	} else cout << "Unable to open file";

	// CUDA:
	

	int size = sizeof(location_item);

	hipMalloc((void **) &d_items, size);

	hipMemcpy(d_items, &items, size, hipMemcpyHostToDevice);

	calculate_distance<<<1,1>>>(d_items);

	test_kernel<<<1,1>>>();

	hipMemcpy(&items, d_items, size, hipMemcpyDeviceToHost);

	hipFree(d_items);


	return 0;
}

