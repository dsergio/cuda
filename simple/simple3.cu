


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <math.h>

using namespace std;

#define _USE_MATH_DEFINES

#define ZRO 737	

#define THREADS_PER_BLOCK 512
#define N 800


struct location_item {
	const char * name;
	double lat;
	double lon;
};

__device__ double toRad(double degree) {
    return degree / 180 * M_PI;
}

__device__ double calculateDistance(double lat1, double long1, double lat2, double long2) {
    double dist;
    dist = sin(toRad(lat1)) * sin(toRad(lat2)) + cos(toRad(lat1)) * cos(toRad(lat2)) * cos(toRad(long1 - long2));
    dist = acos(dist);

		// dist = (6371 * pi * dist) / 180;
		// got dist in radian, no need to change back to degree and convert to rad again.

    dist = 6371 * dist;

    return dist;
}

// test function, do nothing
__global__ void test_kernel(void) {
	printf("do nothing\n");
}

// add integers:
__global__ void add(int *a, int *b, int *c) {
	*c = *a + *b;
	printf("IN ADD: a = %i\n", *a);
}

// calculate_distance:
__global__ void calculate_distance(double *lat, double *lon, double *dist, int n) {

	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < n) {
		double d = calculateDistance(lat[index], lon[index], lat[ZRO], lon[ZRO]);
		dist[index] = d;
	}
	
}


int main(void) {

	printf("test CUDA\n\n");


	// location_item items[N];

	int coords_size = N * sizeof(double);
	cout << "coords_size: " << coords_size << " " << INT_MAX << endl;
	
	double *lat_arr, *lon_arr, *dist_arr;
	lat_arr = (double *) malloc(coords_size);
	lon_arr = (double *) malloc(coords_size);
	dist_arr = (double *) malloc(coords_size);

	double *d_lat_arr, *d_lon_arr, *d_dist_arr;


	string names_arr[N];

	// load data line by line O(n)...
	int i = 0;
	string line;
	ifstream myfile ("wa_cities");
	if (myfile.is_open()) {
		while (getline(myfile, line) && i < N) {
			
			int pos = line.find(";");
			string name = line.substr(0, pos);
			string coords = line.substr(pos + 1, strlen(line.c_str()) - pos);
			int comma = coords.find(",");
			double lat = stod(coords.substr(0, comma));
			double lon = stod(coords.substr(comma + 1, strlen(coords.c_str()) - comma));

			// items[i] = (location_item){name.c_str(), lat, lon};
			lat_arr[i] = lat;
			lon_arr[i] = lon;

			names_arr[i] = name;
			i++;
		}
		myfile.close();
	} else cout << "Unable to open file";



	for (int i = 0; i < N; i++) {
		// printf("i = %d: before CUDA: lat: %lf, name: %s\n", i, lat_arr[i], names_arr[i].c_str());
	}

	// CUDA:
	
	hipMalloc((void **) &d_lat_arr, coords_size);
	hipMalloc((void **) &d_lon_arr, coords_size);
	hipMalloc((void **) &d_dist_arr, coords_size);

	hipMemcpy(d_lat_arr, lat_arr, coords_size, hipMemcpyHostToDevice);
	hipMemcpy(d_lon_arr, lon_arr, coords_size, hipMemcpyHostToDevice);

	calculate_distance<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_lat_arr, d_lon_arr, d_dist_arr, N);

	hipMemcpy(dist_arr, d_dist_arr, coords_size, hipMemcpyDeviceToHost);

	for (int i = 0; i < N; i++) {

		if (dist_arr[i] > 500) {
			cout << "after CUDA:";
			cout << "\tlat: " <<  lat_arr[i] << ", long: " << lon_arr[i];
			cout << "\tdistance (from " << names_arr[ZRO] << " to " << names_arr[i] << "): \t\t" << 
				dist_arr[i] << endl;
		}

	}

	free(lat_arr);
	free(lon_arr);
	free(dist_arr);
	hipFree(d_lat_arr);
	hipFree(d_lon_arr);
	hipFree(d_dist_arr);


	return 0;
}

